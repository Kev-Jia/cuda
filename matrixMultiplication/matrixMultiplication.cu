#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>

// kernel
__global__ void multiplyMatricesKernel(float* d_x, float* d_y, float* d_z, int m, int n, int p)
{
    // indexing variables
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    // thread boundary check
    if(i < p && j < m)
    {
        for(int k = 0; k < n; ++k)
        {
            d_z[j * p + i] += d_x[j * n + k] * d_y[k * p + i];
        }
    }
}

// CUDA error checking
void errorCheck(unsigned int line)
{
    hipError_t hipError_t = hipGetLastError();
    
    if(hipError_t != hipSuccess)
    {
        printf("CUDA error in line %u in file %s: %s\n", line - 1, __FILE__, hipGetErrorString(hipError_t));
        exit(EXIT_FAILURE);
    }
}

// host function containing kernel call
void multiplyMatrices(float* x, float* y, float* z, int m, int n, int p)
{
    dim3 numOfBlocks(ceil(p / 32.0), ceil(m / 32.0), 1);
    dim3 numOfThreads(32, 32, 1);
    
    size_t bytes_x = m * n * sizeof(float);
    size_t bytes_y = n * p * sizeof(float);
    size_t bytes_z = m * p * sizeof(float);

    float* d_x;
    float* d_y;
    float* d_z;

    hipMalloc((void**) &d_x, bytes_x);
    errorCheck(__LINE__);
    hipMalloc((void**) &d_y, bytes_y);
    errorCheck(__LINE__);
    hipMalloc((void**) &d_z, bytes_z);
    errorCheck(__LINE__);

    hipMemcpy(d_x, x, bytes_x, hipMemcpyHostToDevice);
    errorCheck(__LINE__);
    hipMemcpy(d_y, y, bytes_y, hipMemcpyHostToDevice);
    errorCheck(__LINE__);

    multiplyMatricesKernel<<<numOfBlocks, numOfThreads>>>(d_x, d_y, d_z, m, n, p);
    errorCheck(__LINE__);

    hipMemcpy(z, d_z, bytes_z, hipMemcpyDeviceToHost);
    errorCheck(__LINE__);

    hipFree(d_x);
    errorCheck(__LINE__);
    hipFree(d_y);
    errorCheck(__LINE__);
    hipFree(d_z);
    errorCheck(__LINE__);
}

int main()
{
    struct timespec start, end;

    clock_gettime(CLOCK_REALTIME, &start);

    srand(time(NULL));

    size_t m = rand() % 257 + 3840;
    size_t n = rand() % 257 + 3840;
    size_t p = rand() % 257 + 3840;

    float* x = (float*) malloc(m * n * sizeof(float));
    float* y = (float*) malloc(n * p * sizeof(float));
    float* z = (float*) malloc(m * p * sizeof(float));

    for(int i = 0; i < m * n; ++i)
    {
        x[i] = rand() % 129 - 64;
    }

    for(int i = 0; i < n * p; ++i)
    {
        y[i] = rand() % 129 - 64;
    }
    
    // do matrix multiplication
    multiplyMatrices(x, y, z, m, n, p);
    
    clock_gettime(CLOCK_REALTIME, &end);

    time_t execTime = (end.tv_sec - start.tv_sec) * 1000000 + (end.tv_nsec - start.tv_nsec) / 1000;

    printf("Execution time: %d microseconds.", execTime);

    return 0;
}
