#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>



// CUDA error checking
void errorCheck(unsigned int line)
{
    hipError_t hipError_t = hipGetLastError();

    if(hipError_t != hipSuccess)
    {
        printf("CUDA error in line %u in file %s: %s\n", line - 1, __FILE__, hipGetErrorString(hipError_t));
        exit(EXIT_FAILURE);
    }
}

