#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>

// kernel
__global__ void convolution_2D_Kernel(float* d_m, float* d_mask, float* d_n, size_t a, size_t b, size_t maskWidth)
{
    // indexing variables
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    int m_row = j - maskWidth / 2;
    int m_col = i - maskWidth / 2;

    // thread boundary check
    if(i < b && j < a)
    {
        for(int k = 0; k < maskWidth; ++k)
        {
            for(int l = 0; l < maskWidth; ++l)
            {
                if(m_row + l >= 0 && m_row + l < a && m_col + k >= 0 && m_col + k < b)
                {
                    d_n[j * b + i] += d_m[(m_row + l) * b + m_col + k] * d_mask[l * maskWidth + k];
                }
            }
        }
    }
}

// CUDA error checking
void errorCheck(unsigned int line)
{
    hipError_t hipError_t = hipGetLastError();

    if(hipError_t != hipSuccess)
    {
        printf("CUDA error in line %u in file %s: %s\n", line - 1, __FILE__, hipGetErrorString(hipError_t));
        exit(EXIT_FAILURE);
    }
}

// host function containing kernel call
void convolution_2D(float* m, float* mask, float* n, size_t a, size_t b, size_t maskWidth)
{
    dim3 numOfBlocks(ceil(b / 32.0), ceil(a / 32.0), 1);
    dim3 numOfThreads(32, 32, 1);
    
    size_t bytes_m = a * b * sizeof(float);
    size_t bytes_mask = maskWidth * maskWidth * sizeof(float);
    size_t bytes_n = a * b * sizeof(float);

    float* d_m;
    float* d_mask;
    float* d_n;

    hipMalloc((void**) &d_m, bytes_m);
    errorCheck(__LINE__);
    hipMalloc((void**) &d_mask, bytes_mask);
    errorCheck(__LINE__);
    hipMalloc((void**) &d_n, bytes_n);
    errorCheck(__LINE__);

    hipMemcpy(d_m, m, bytes_m, hipMemcpyHostToDevice);
    errorCheck(__LINE__);
    hipMemcpy(d_mask, mask, bytes_mask, hipMemcpyHostToDevice);
    errorCheck(__LINE__);

    convolution_2D_Kernel<<<numOfBlocks, numOfThreads>>>(d_m, d_mask, d_n, a, b, maskWidth);
    errorCheck(__LINE__);

    hipMemcpy(n, d_n, bytes_n, hipMemcpyDeviceToHost);
    errorCheck(__LINE__);

    hipFree(d_m);
    errorCheck(__LINE__);
    hipFree(d_mask);
    errorCheck(__LINE__);
    hipFree(d_n);
    errorCheck(__LINE__);
}

int main()
{
    struct timespec start, end;

    srand(time(NULL));

    size_t a = rand() % 257 + 3840;
    size_t b = rand() % 257 + 3840;
    size_t maskWidth = 11;

    float* m = (float*) malloc(a * b * sizeof(float));
    float* mask = (float*) malloc(maskWidth * maskWidth * sizeof(float));
    float* n = (float*) malloc(a * b * sizeof(float));

    for(int i = 0; i < a * b; ++i)
    {
        m[i] = rand() % 129 - 64;
    }

    for(int j = 0; j < maskWidth * maskWidth; ++j)
    {
        mask[j] = rand() % 1001 / 1000.0;
    }

    clock_gettime(CLOCK_REALTIME, &start);
    
    // do convolution
    convolution_2D(m, mask, n, a, b, maskWidth);

    clock_gettime(CLOCK_REALTIME, &end);

    time_t execTime = (end.tv_sec - start.tv_sec) * 1000000 + (end.tv_nsec - start.tv_nsec) / 1000;

    printf("Execution time: %d microseconds.", execTime);

    return 0;
}
